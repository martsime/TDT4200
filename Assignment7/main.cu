#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
extern "C" {
    #include "libs/bitmap.h"
}

#define ERROR_EXIT -1

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int GPU = 1;
const int CPU = 0;

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

int const sobelYFilter[] = {-1, -2, -1,
                             0,  0,  0,
                             1,  2,  1};
float const sobelYFilterFactor = (float) 1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2,  0, -2,
                            -1,  0, -1 , 0};
float const sobelXFilterFactor = (float) 1.0;


int const laplacian1Filter[] = {  -1,  -4,  -1,
                                 -4,  20,  -4,
                                 -1,  -4,  -1};

float const laplacian1FilterFactor = (float) 1.0;

int const laplacian2Filter[] = { 0,  1,  0,
                                 1, -4,  1,
                                 0,  1,  0};
float const laplacian2FilterFactor = (float) 1.0;

int const laplacian3Filter[] = { -1,  -1,  -1,
                                  -1,   8,  -1,
                                  -1,  -1,  -1};
float const laplacian3FilterFactor = (float) 1.0;


//Bonus Filter:

int const gaussianFilter[] = { 1,  4,  6,  4, 1,
                               4, 16, 24, 16, 4,
                               6, 24, 36, 24, 6,
                               4, 16, 24, 16, 4,
                               1,  4,  6,  4, 1 };

float const gaussianFilterFactor = (float) 1.0 / 256.0;


// Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[y][x] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[y][x] = 0;
      }
    }
  }
}

__global__ void applyFilterCuda(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) { 
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int ix = blockDim.x * blockIdx.x + threadIdx.x;

    // Boundary check
    if (ix < 0 || ix >= (int) width || iy < 0 || iy >= (int) height)
        return;

    unsigned int const filterCenter = (filterDim / 2);
    int aggregate = 0;
    for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
            int nkx = filterDim - 1 - kx;
            int y = iy + (ky - filterCenter);
            int x = ix + (kx - filterCenter);
            if (x >= 0 && x < (int) width && y >=0 && y < (int) height) {
                aggregate += in[y * width + x] * filter[nky * filterDim + nkx];
            }
        }
    }
    aggregate *= filterFactor;
  if (aggregate > 0) {
    out[iy * width + ix] = (aggregate > 255) ? 255 : aggregate;
  } else {
    out[iy * width + ix] = 0;
  }
}

void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  int ret = 0;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hi:";
  {
    char *endptr;
    int c;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        return 0;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          return ERROR_EXIT;
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    return ERROR_EXIT;
  }
  input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(input, argv[optind], strlen(argv[optind]));
  optind++;

  output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(output, argv[optind], strlen(argv[optind]));
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    return ERROR_EXIT;
  }


  // Create a single color channel image. It is easier to work just with one color
  bmpImageChannel *imageChannel = newBmpImageChannel(image->width, image->height);
  if (imageChannel == NULL) {
    fprintf(stderr, "Could not allocate new image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(imageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }

  int imageSize = image->width * image->height * sizeof(unsigned char);
  unsigned char *cudaRawInImage;
  unsigned char *cudaRawOutImage;
  if (GPU) {
      cudaErrorCheck(hipMalloc(&cudaRawInImage, imageSize));
      cudaErrorCheck(hipMalloc(&cudaRawOutImage, imageSize));
      cudaErrorCheck(hipMemcpy(cudaRawInImage, imageChannel->rawdata, imageSize, hipMemcpyHostToDevice));
  }

  // Specify which filter to use
  int *filter = (int *) laplacian1Filter;
  int filterSize = sizeof(laplacian1Filter);
  unsigned int filterDim = 3;
  float filterFactor = laplacian1FilterFactor;

  // Copy the filter to device
  int *cudaFilter;
  if (GPU) {
      cudaErrorCheck(hipMalloc(&cudaFilter, filterSize));
      cudaErrorCheck(hipMemcpy(cudaFilter, filter, filterSize, hipMemcpyHostToDevice));
  }

  dim3 threadsPerBlock(8, 8);
  dim3 numBlocks(imageChannel->width / threadsPerBlock.x + 1, imageChannel->height / threadsPerBlock.y + 1);


  //Here we do the actual computation!
  // imageChannel->data is a 2-dimensional array of unsigned char which is accessed row first ([y][x])
  bmpImageChannel *processImageChannel;
  if (CPU) {
    processImageChannel = newBmpImageChannel(imageChannel->width, imageChannel->height);
  }
  for (unsigned int i = 0; i < iterations; i ++) {
    if (CPU) {
        applyFilter(processImageChannel->data,
                    imageChannel->data,
                    imageChannel->width,
                    imageChannel->height,
                    filter, filterDim, filterFactor
     //               (int *)laplacian2Filter, 3, laplacian2FilterFactor
     //               (int *)laplacian3Filter, 3, laplacian3FilterFactor
     //               (int *)gaussianFilter, 5, gaussianFilterFactor
                    );
        // Swap imageChannels for cpu
        swapBmpImageChannels(imageChannel, processImageChannel);
    }

    if (GPU) {
        applyFilterCuda<<<numBlocks, threadsPerBlock>>>(cudaRawOutImage, cudaRawInImage, imageChannel->width, imageChannel->height, cudaFilter, filterDim, filterFactor);

        // Swap the data pointers for gpu
        unsigned char *tmp = cudaRawInImage;
        cudaRawInImage = cudaRawOutImage;
        cudaRawOutImage = tmp;
    }

  }
  if (CPU) {
    freeBmpImageChannel(processImageChannel);
  }
  bmpImageChannel *cudaResultImageChannel;
  if (GPU) { 
      cudaResultImageChannel = newBmpImageChannel(imageChannel->width, imageChannel->height);
        
      cudaErrorCheck(hipMemcpy(cudaResultImageChannel->rawdata, cudaRawInImage, imageSize, hipMemcpyDeviceToHost));

      // Free cuda memory
      cudaErrorCheck(hipFree(cudaRawInImage));
      cudaErrorCheck(hipFree(cudaRawOutImage));
      cudaErrorCheck(hipFree(cudaFilter)); 
  }

  if (GPU && CPU) {
      for (unsigned int y = 0; y < imageChannel->height; y++) {
        for (unsigned int x = 0; x < imageChannel->width; x++) {
          if (imageChannel->data[y][x] != cudaResultImageChannel->data[y][x]) {
              unsigned char cpu = imageChannel->data[y][x];
              unsigned char gpu = cudaResultImageChannel->data[y][x];
              printf("cpu: %d != gpu: %d at index (%d, %d)\n", cpu, gpu, x, y);
          }
        }
      }
  }

  // Map our single color image back to a normal BMP image with 3 color channels
  // mapEqual puts the color value on all three channels the same way
  // other mapping functions are mapRed, mapGreen, mapBlue
  if (mapImageChannel(image, imageChannel, mapEqual) != 0) {
    fprintf(stderr, "Could not map image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }
  freeBmpImageChannel(imageChannel);
  if (GPU) {
    freeBmpImageChannel(cudaResultImageChannel);
  }

  //Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    return ERROR_EXIT;
  };

  ret = 0;
  if (input)
    free(input);
  if (output)
    free(output);
  return ret;
};
